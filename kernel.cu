#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>
#include <iostream>

#define STATES_NUM 2000
#define OBSERVATIONS_NUM 2000
#define RANDOMIZER_ACCURACY 10000
#define BLOCK_SIZE 1024

//����� ������ == 1, �.�. ����� ��������� �� ����� � ������ (��� BLOCK_SIZE == 1024)
const int BLOCK_NUM = STATES_NUM / BLOCK_SIZE + 1;

double min4(double a, double b, double c, double d);
void fillInitialDistribution(double* initialDistribution);
void generateObservations(int* observations);
void generateTransitions(double** Transitions);
void generateEmissions(double** Emissions);
int* viterbi(double initialDistribution[], int observations[], double** Transitions, double** Emissions);
int* viterbiGPU(double initialDistribution[], int observations[], double** Transitions, double** Emissions);

int main()
{
	//������ ��������� ������������
	//(����������� ����, � ��������� ������ ������� ��������� ���� S_i)
	double* initialDistribution = new double[STATES_NUM];

	//������������������ ����������
	int* observations = new int[OBSERVATIONS_NUM];

	//������� ������������ �������� �� i-�� ��������� � j-���
	double** Transitions = new double* [STATES_NUM];
	for (int i = 0; i < STATES_NUM; i++)
		Transitions[i] = new double[STATES_NUM];

	//������� ������������ ���������� O_j �� ��������� S_i
	double** Emissions = new double* [STATES_NUM];
	for (int i = 0; i < STATES_NUM; i++)
		Emissions[i] = new double[STATES_NUM];

	//��� ���������� ��������, ��������� ������������� ���� �� ��� ������,
	//������� ������� ��� ���������� (�� �����)
	fillInitialDistribution(initialDistribution);

	generateObservations(observations);

	//����������� ������ �������������� ������� ���������
	generateTransitions(Transitions);

	//� �������������� �� �������� ������� �������
	generateEmissions(Emissions);

	clock_t start, end;
	
	start = clock();
	viterbi(initialDistribution, observations, Transitions, Emissions);
	end = clock();
	std::cout << "Viterbi elapsed time: " << double(end - start) / CLOCKS_PER_SEC << " sec\n";

	start = clock();
	viterbiGPU(initialDistribution, observations, Transitions, Emissions);
	end = clock();
	std::cout << "ViterbiGPU elapsed time: " << double(end - start) / CLOCKS_PER_SEC << " sec\n";

	for (int i = 0; i < STATES_NUM; i++)
		delete[]Transitions[i];
	delete[]Transitions;

	for (int i = 0; i < STATES_NUM; i++)
		delete[]Emissions[i];
	delete[]Emissions;

	delete[]initialDistribution;
	delete[]observations;

	return 0;
}

double min4(double a, double b, double c, double d)
{
	double temp[4] = { a, b, c, d };
	double min = DBL_MAX;
	for (int i = 0; i < 4; i++)
		if (temp[i] < min)
			min = temp[i];

	return min;
}

//(��, ���, ��������, ���������� ������, �� � �� �������� ������ �����)
void generateTransitions(double** Transitions)
{
	double n = 1.0 / STATES_NUM;

	for (int i = 0; i < STATES_NUM; i++)
		for (int j = 0; j < OBSERVATIONS_NUM; j++)
			Transitions[i][j] = n;

	srand(time(NULL));

	for (int i = 0; i < STATES_NUM; i++)
	{
		for (int j = 0; j < STATES_NUM; j++)
		{
			int randomIndex = -1;
			do
			{
				randomIndex = rand() % STATES_NUM;
			} while (randomIndex == i || randomIndex == j);

			double maxDelta = min4(Transitions[i][randomIndex],
				Transitions[STATES_NUM - i - 1][STATES_NUM - randomIndex - 1],
				1 - Transitions[i][STATES_NUM - randomIndex - 1],
				1 - Transitions[STATES_NUM - i - 1][randomIndex])
				/ 2;

			//delta - ��������� ���� �� 0 �� maxDelta
			double delta = (double)(rand() % (RANDOMIZER_ACCURACY + 1)) / RANDOMIZER_ACCURACY * maxDelta;
			Transitions[i][j] -= delta;
			Transitions[STATES_NUM - i - 1][STATES_NUM - j - 1] -= delta;
			Transitions[i][STATES_NUM - j - 1] += delta;
			Transitions[STATES_NUM - i - 1][j] += delta;
		}
	}
}

void generateEmissions(double** Emissions)
{
	double n = 1.0 / STATES_NUM;

	for (int i = 0; i < STATES_NUM; i++)
		for (int j = 0; j < OBSERVATIONS_NUM; j++)
			Emissions[i][j] = n;

	srand(time(NULL));

	for (int i = 0; i < STATES_NUM; i++)
	{
		for (int j = 0; j < OBSERVATIONS_NUM; j++)
		{
			int randomIndex;
			do
			{
				randomIndex = rand() % STATES_NUM;
			} while (randomIndex == j);

			double min = 0;
			if (Emissions[i][j] < Emissions[i][randomIndex])
				min = Emissions[i][j];
			else
				min = Emissions[i][randomIndex];

			double maxDelta = min / 2;

			//delta - ��������� ���� �� 0 �� maxDelta
			double delta = (double)(rand() % (RANDOMIZER_ACCURACY + 1)) / RANDOMIZER_ACCURACY * maxDelta;
			Emissions[i][j] -= delta;
			Emissions[i][randomIndex] += delta;
		}
	}
}

void generateObservations(int* observations)
{
	srand(time(NULL));
	for (int i = 0; i < OBSERVATIONS_NUM; i++)
		observations[i] = rand() % 2;
}

void fillInitialDistribution(double* initialDistribution)
{
	double n = 1.0 / STATES_NUM;
	double sum = 0;
	for (int i = 0; i < STATES_NUM - 1; i++)
	{
		initialDistribution[i] = n;
		sum += n;
	}
	//��� ������� ����� ������������� �����������, ������� ������� ���
	initialDistribution[STATES_NUM - 1] = 1 - sum;
}

//������ ����������� ������, �� ������������ �� �����
int* viterbi(double initialDistribution[], int observations[], double** Transitions, double** Emissions)
{
	//������� ������������ ����, ��� �� j-��� ���� �� ��������� � ��������� S_i
	double** MState = new double* [STATES_NUM];
	for (int i = 0; i < STATES_NUM; i++)
		MState[i] = new double[OBSERVATIONS_NUM];

	//������� �������� �������� ��������� ��������� �� j - 1 ����
	int** MIndex = new int* [STATES_NUM];
	for (int i = 0; i < STATES_NUM; i++)
		MIndex[i] = new int[OBSERVATIONS_NUM];

	//��������� ������ ������� �� ������ ��������� ������
	for (int i = 0; i < STATES_NUM; i++)
	{
		MState[i][0] = initialDistribution[i] * Emissions[i][observations[i]];
		MIndex[i][0] = 0;
	}

	//��������� �����������
	for (int i = 1; i < OBSERVATIONS_NUM; i++) {
		for (int j = 0; j < STATES_NUM; j++) {
			//���� ������, ��� ������� ��������������� func
			int indMax = -1;
			for (int k = 0; k < STATES_NUM; k++)
			{
				double func = MState[k][i - 1] * Transitions[k][j] * Emissions[j][observations[i]];
				if (MState[j][i] < func)
				{
					MState[j][i] = func;
					indMax = k;
				}
			}

			MIndex[j][i] = indMax;
		}
	}

	double max = -1;
	int* result = new int[OBSERVATIONS_NUM];

	//������ ������ ���������� ���������
	for (int i = 0; i < STATES_NUM; i++)
	{
		if (MState[i][OBSERVATIONS_NUM - 1] > max)
		{
			max = MState[i][OBSERVATIONS_NUM - 1];
			result[OBSERVATIONS_NUM - 1] = i;
		}
	}

	//��������� ���������
	for (int i = OBSERVATIONS_NUM - 2; i > 0; i--)
		result[i] = MIndex[result[i + 1]][i + 1];

	for (int i = 0; i < STATES_NUM; i++)
		delete[]MState[i];
	delete[]MState;

	for (int i = 0; i < STATES_NUM; i++)
		delete[]MIndex[i];
	delete[]MIndex;

	return result;
}

//������ ��������� �� �����( � ������

__global__
void viterbiGPU_forward(double* MState, double* Transitions, double* Emissions, int* Observations, int i) 
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if (k >= STATES_NUM)
		return;

	MState[i * STATES_NUM + k] = -1;

	for (int j = 0; j < STATES_NUM; j++) 
	{
		double func = MState[(i - 1) * STATES_NUM + j] * Transitions[j * STATES_NUM + k]
					* Emissions[k * 2 + Observations[i]];

		if (MState[i * STATES_NUM + k] < func) 
			MState[i * STATES_NUM + k] = func;
	}
}

__global__
void viterbiGPU_back(double* MState, double* Transitions, double* MIndex, int i) 
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if (k >= STATES_NUM)
		return;

	double max = -1;
	for (int j = 0; j < STATES_NUM; j++) 
	{
		double func = MState[i * STATES_NUM + j] * Transitions[j * STATES_NUM + k];
		if (max < func) 
		{
			max = func;
			MIndex[i * STATES_NUM + k] = j;
		}
	}
}

void copyMatrixFromDevice(double** matrix, double* deviceMatrix, int ROWS_NUM, int COLUMN_NUM) 
{
	double** temp = new double *[ROWS_NUM * COLUMN_NUM];

	hipMemcpy(temp, deviceMatrix, ROWS_NUM * COLUMN_NUM * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < ROWS_NUM; i++)
		memcpy(matrix[i], temp + i * COLUMN_NUM, COLUMN_NUM * sizeof(double));
	
	delete[] temp;
}

void copyMatrixToDevice(double **matrix, double *deviceMatrix, int ROWS_NUM, int COLUMN_NUM)
{
	double** temp = new double* [ROWS_NUM * COLUMN_NUM];

	for (int i = 0; i < ROWS_NUM; i++)
		memcpy(temp + i * COLUMN_NUM, matrix[i], COLUMN_NUM * sizeof(double));
	hipMemcpy(deviceMatrix, temp, ROWS_NUM * COLUMN_NUM * sizeof(double), hipMemcpyHostToDevice);

	delete[] temp;
}


int* viterbiGPU(double initialDistribution[], int observations[], double** Transitions, double** Emissions)
{
	double** MState = new double* [STATES_NUM];
	for (int i = 0; i < STATES_NUM; i++)
		MState[i] = new double[OBSERVATIONS_NUM];

	for (int i = 0; i < STATES_NUM; i++) 
		MState[0][i] = initialDistribution[i] * Emissions[i][observations[0]];

	double** MIndex = new double* [OBSERVATIONS_NUM];
	for (int i = 0; i < OBSERVATIONS_NUM; i++)
		MIndex[i] = new double[STATES_NUM];

	int* deviceObservations;
		hipMalloc(&deviceObservations, OBSERVATIONS_NUM * sizeof(int));

	double* deviceTransitions;
		hipMalloc(&deviceTransitions, STATES_NUM * STATES_NUM * sizeof(double));

	double* deviceEmissions;
		hipMalloc(&deviceEmissions, STATES_NUM * STATES_NUM * sizeof(double));

	double* deviceMState;
		hipMalloc(&deviceMState, STATES_NUM * OBSERVATIONS_NUM * sizeof(double));

	double* deviceMIndex;
		hipMalloc(&deviceMIndex, STATES_NUM * OBSERVATIONS_NUM * sizeof(double));

	hipMemcpy(deviceObservations, observations, OBSERVATIONS_NUM * sizeof(int), hipMemcpyHostToDevice);
	copyMatrixToDevice(Transitions, deviceTransitions, STATES_NUM, STATES_NUM);
	copyMatrixToDevice(Emissions, deviceEmissions, STATES_NUM, STATES_NUM);
	copyMatrixToDevice(MState, deviceMState, STATES_NUM, OBSERVATIONS_NUM);
	copyMatrixToDevice(MIndex, deviceMIndex, STATES_NUM, OBSERVATIONS_NUM);

	hipDeviceSynchronize();

	for (int i = 1; i < OBSERVATIONS_NUM; i++) 
	{
		viterbiGPU_forward<<<BLOCK_NUM, BLOCK_SIZE>>>
			(deviceMState, deviceTransitions, deviceEmissions, deviceObservations, i);
		hipDeviceSynchronize();
	}
	for (int i = 0; i < OBSERVATIONS_NUM; i++) 
		viterbiGPU_back<<<BLOCK_NUM, BLOCK_SIZE>>>
			(deviceMState, deviceTransitions, deviceMIndex, i);

	copyMatrixFromDevice(MState, deviceMState, OBSERVATIONS_NUM, STATES_NUM);
	copyMatrixFromDevice(MIndex, deviceMIndex, OBSERVATIONS_NUM, STATES_NUM);

	hipFree(deviceMIndex);
	hipFree(deviceMState);
	hipFree(deviceTransitions);
	hipFree(deviceEmissions);
	hipFree(deviceObservations);

	double max = -1;
	int* result = new int[OBSERVATIONS_NUM];

	for (int i = 0; i < STATES_NUM; i++)
		if (MState[OBSERVATIONS_NUM - 1][i] > max) 
		{
			max = MState[OBSERVATIONS_NUM - 1][i];
			result[OBSERVATIONS_NUM - 1] = i;
		}


	for (int i = OBSERVATIONS_NUM - 2; i >= 0; i--)
		result[i] = (int)MIndex[i + 1][result[i + 1]];

	for (int i = 0; i < OBSERVATIONS_NUM; i++) 
		delete MIndex[i];
	delete[] MIndex;

	for (int i = 0; i < OBSERVATIONS_NUM; i++)
		delete MState[i];
	delete[] MState;

	return result;
}